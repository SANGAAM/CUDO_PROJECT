#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<hip/hip_runtime.h>

#include <stdio.h>

__global__ void addKernel(int * dev_a)
{
    int i = threadIdx.x;
	dev_a[i] = dev_a[i] + 15;
}

int main()
{
	const int size = 6;
	int a[size][size];
	int b[size][size] = {0};
	int i = 0, j = 0;
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
			scanf("%d", &a[i][j]);

	int *dev_a;
	int t = size*size*sizeof(int);
	hipMalloc((void**)&dev_a, t);
	hipMemcpy(dev_a, a, t, hipMemcpyHostToDevice);
	addKernel << <1, size*size >> >(dev_a);
	hipMemcpy(b, dev_a, t, hipMemcpyDeviceToHost);
	for (i = 0; i < size; i++){
		for (j = 0; j < size; j++){
			printf("%d ", b[i][j]);
		}
		printf("\n");
	}

    return 0;
}



