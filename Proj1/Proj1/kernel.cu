#include <stdio.h>

#include <hip/hip_runtime.h>


__global__ void addKernel(int *c, int *a, int *b);

int main()
{
    const int arraySize = 5;
	int a[arraySize];
	 int b[arraySize];
	 int c[arraySize] = { 0 };
	int *dev_c, *dev_a, *dev_b;
	int i;
	for (i = 0; i < arraySize; i++)
		scanf("%d", &a[i]);
	for (i = 0; i < arraySize; i++)
		scanf("%d", &b[i]);

	hipMalloc((void**)&dev_c, arraySize*sizeof(int));
	hipMalloc((void**)&dev_a, arraySize*sizeof(int));
	hipMalloc((void**)&dev_b, arraySize*sizeof(int));
	hipMemcpy(dev_a, a, arraySize*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, arraySize*sizeof(int), hipMemcpyHostToDevice);
	addKernel<<<1, arraySize >>>(dev_c,dev_a,dev_b);
	hipMemcpy(&c, dev_c, arraySize*sizeof(int), hipMemcpyDeviceToHost);

    printf("{%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

   
    
    return 0;
}
__global__ void addKernel(int *c, int *a,int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
	//printf("%d", c[i]);
}
